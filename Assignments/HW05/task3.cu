#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include "vscale.cuh"

int main(int argc, char* argv[]) {
    // Open the output file in truncation mode to ensure a clean start.
    std::ofstream outFile("results.txt", std::ofstream::out | std::ofstream::trunc);
    if (!outFile) {
        std::cerr << "Error opening output file." << std::endl;
        return 1;
    }
    // Write header line.
    outFile << "n time_512 time_16 first_512 last_512 first_16 last_16\n";

    // Seed the random number generator.
    std::srand(static_cast<unsigned>(std::time(nullptr)));

    // Loop over exponents 10 to 29 (n = 2^exp).
    for (int exp = 10; exp <= 29; exp++) {
        unsigned int n = 1u << exp;  // n = 2^exp

        // Allocate and initialize host arrays.
        float* hA = new float[n];
        float* hB = new float[n];
        for (unsigned int i = 0; i < n; ++i) {
            // hA: random float in [-10, 10]
            hA[i] = (static_cast<float>(std::rand()) / RAND_MAX) * 20.0f - 10.0f;
            // hB: random float in [0, 1]
            hB[i] = static_cast<float>(std::rand()) / RAND_MAX;
        }

        // Allocate device memory for a and b.
        float *dA, *dB;
        hipMalloc(&dA, n * sizeof(float));
        hipMalloc(&dB, n * sizeof(float));

        // Copy hA and hB to device.
        hipMemcpy(dA, hA, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice);

        // ---- Experiment 1: Using 512 threads per block ----
        unsigned int threads512 = 512;
        unsigned int numBlocks512 = (n + threads512 - 1) / threads512;

        hipEvent_t start512, stop512;
        hipEventCreate(&start512);
        hipEventCreate(&stop512);

        hipEventRecord(start512);
        vscale<<<numBlocks512, threads512>>>(dA, dB, n);
        hipEventRecord(stop512);
        hipEventSynchronize(stop512);
        float time512 = 0.0f;
        hipEventElapsedTime(&time512, start512, stop512);

        // Copy results from device.
        float *result512 = new float[n];
        hipMemcpy(result512, dB, n * sizeof(float), hipMemcpyDeviceToHost);
        float first512 = result512[0];
        float last512 = result512[n - 1];

        hipEventDestroy(start512);
        hipEventDestroy(stop512);

        // ---- Experiment 2: Using 16 threads per block ----
        // Restore original hB values into device array dB.
        hipMemcpy(dB, hB, n * sizeof(float), hipMemcpyHostToDevice);

        unsigned int threads16 = 16;
        unsigned int numBlocks16 = (n + threads16 - 1) / threads16;

        hipEvent_t start16, stop16;
        hipEventCreate(&start16);
        hipEventCreate(&stop16);

        hipEventRecord(start16);
        vscale<<<numBlocks16, threads16>>>(dA, dB, n);
        hipEventRecord(stop16);
        hipEventSynchronize(stop16);
        float time16 = 0.0f;
        hipEventElapsedTime(&time16, start16, stop16);

        float *result16 = new float[n];
        hipMemcpy(result16, dB, n * sizeof(float), hipMemcpyDeviceToHost);
        float first16 = result16[0];
        float last16 = result16[n - 1];

        hipEventDestroy(start16);
        hipEventDestroy(stop16);

        // Write one clean line of results for this n.
        outFile << n << " " << time512 << " " << time16 << " " 
                << first512 << " " << last512 << " " 
                << first16 << " " << last16 << "\n";

        // Cleanup for this iteration.
        hipFree(dA);
        hipFree(dB);
        delete[] hA;
        delete[] hB;
        delete[] result512;
        delete[] result16;
    }

    outFile.close();
    return 0;
}
